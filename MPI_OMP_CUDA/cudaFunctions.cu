#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
extern "C"{
#include "myProto.h"
}

__device__ int is_consevative(char seq1_char , char seq_other_char);
__device__ int is_semi_consevative(char seq1_char , char seq_other_char);

__global__ void alignment_score(double* array_score,int numElements,char *seq1,char *seq2,int offset,int hypen,double w1,double w2,double w3,double w4)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(numElements > i)
    {
		if(i < hypen)
		{
			if(seq1[i+offset] ==  seq2[i])
	        {
	            array_score[i] = w1;
	        }
	        // Check Consevative
	        else if(is_consevative(seq1[i+offset],seq2[i]))
	        {
	            array_score[i] = -w2;
	
	        }
	        // Check Semi Consevative
	        else if(is_semi_consevative(seq1[i+offset],seq2[i]))
	        {
	            array_score[i] = -w3;
	        }
	        else
	        {
	            array_score[i] = -w4;
	        }
		}
		else if(i > hypen)
		{
			if(seq1[i+offset] ==  seq2[i-1])
	        {
	            array_score[i] = w1;
	        }
	        // Check Consevative
	        else if(is_consevative(seq1[i+offset],seq2[i-1]))
	        {
	            array_score[i] = -w2;
	
	        }
	        // Check Semi Consevative
	        else if(is_semi_consevative(seq1[i+offset],seq2[i-1]))
	        {
	            array_score[i] = -w3;
	        }
	        else
	        {
	            array_score[i] = -w4;
	        }
		
		}
		else
		{
			array_score[i] = -w4;
		}
    }
}
extern "C"
void cuda_alignment_score(double* array_score,char *seq1,int len_seq1,char *seq2,int len_seq2,int offset,int hypen,double w1,double w2,double w3,double w4)
{
	// Error code to check return values for CUDA calls
     hipError_t err = hipSuccess;
     size_t size1 = len_seq1 * sizeof(char);
     size_t size2 = len_seq2 * sizeof(char);
     size_t size_score = (len_seq2+1) * sizeof(double);

    // Allocate memory on GPU to copy the data from the host
      char *d_seq1;
      char *d_seq2;
      double *d_score;
      err = hipMalloc((void **)&d_seq1, size1);
    	if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }
      err = hipMalloc((void **)&d_seq2, size2);
    	if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }
      err = hipMalloc((void **)&d_score, size_score);
          	if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }
        
        
      // Copy data from host to the GPU memory
    err = hipMemcpy(d_seq1, seq1, size1, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }
    err = hipMemcpy(d_seq2, seq2, size2, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }
   	err = hipMemcpy(d_score,array_score, size_score, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }
    
	// Calculates how many blocks to use based on the size of the arrays
    // Launch the Kernel
    int threadsPerBlock = 1024;
    int blocksPerGrid = (len_seq2 + threadsPerBlock - 1) / threadsPerBlock;
	alignment_score<<<blocksPerGrid, threadsPerBlock>>>(d_score,len_seq2+1,d_seq1,d_seq2,offset,hypen,w1,w2,w3,w4);
    
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }
    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(array_score, d_score, size_score, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Free allocated memory on GPU
    if (hipFree(d_seq1) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
      if (hipFree(d_seq2) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
          if (hipFree(d_score) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
             
}
__device__ int is_consevative(char seq1_char , char seq_other_char)
{
	char conservative_groups[CONSERVATIVE_GROUP_NUMBER][CONSERVATIVE_GROUP_NUMBER_CHARS] = 
		{"NDEQ","NEQK","STA","MILV","QHRK","NHQK","FYW","HY","MILF"};
    int y;
    int x;
        for(y=0; y < CONSERVATIVE_GROUP_NUMBER;y++)
    {
        int is_char1 = 0;
        int is_char2 = 0;
        for(x=0;x<CONSERVATIVE_GROUP_NUMBER_CHARS;x++)
        {
            if(seq1_char == conservative_groups[y][x])
            {
                is_char1 = 1;
            }
            if(seq_other_char == conservative_groups[y][x])
            {
                is_char2 = 1;
            }
        }
       if (is_char1 && is_char2)
       {
           return 1;
       }
    }
    return 0;
}

__device__ int is_semi_consevative(char seq1_char , char seq_other_char)
{
	char semi_groups[SEMI_CONSERVATIVE_GROUP_NUMBER][SEMI_CONSERVATIVE_GROUP_NUMBER_CHARS] =
		{"SAG","ATV","CSA","SGND","STPA","STNK","NEQHRK","NDEQHK","SNDEQK","HFY","FVLIM"};
    int y;
    int x;
        for(y=0; y < SEMI_CONSERVATIVE_GROUP_NUMBER;y++)
    {
        int is_char1 = 0;
        int is_char2 = 0;
        for(x=0;x<SEMI_CONSERVATIVE_GROUP_NUMBER_CHARS;x++)
        {
            if(seq1_char == semi_groups[y][x])
            {
                is_char1 = 1;
            }
            if(seq_other_char == semi_groups[y][x])
            {
                is_char2 = 1;
            }
        }
       if (is_char1 && is_char2)
       {
           return 1;
       }

    }
    return 0;
}

